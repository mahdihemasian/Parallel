#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "bmm.h"

#define tx threadIdx.x
#define ty threadIdx.y

#define bx blockIdx.x
#define by blockIdx.y

// TILEX and TILEY are used to set the number of threads in a CUDA block 
#define TILEX 32
#define TILEY 16

// you may define other parameters here!
// it's lower than tilex and tiley or bigger than both of them
#define TILE 128
// you may define other macros here!
// you may define other functions here!

dim3 getDimGrid(const int m, const int n) {
	dim3 dimGrid(n / TILEX, n / TILEY);
	return dimGrid;
}
dim3 getDimBlock(const int m, const int n) {
	dim3 dimBlock(TILEX, TILEY);
	return dimBlock;
}

__global__ void kernelFunc(float* ad, float* bd, float* cd, const int m, const int n) {

	__shared__ float mad[TILEY][TILE];
	__shared__ float mbd[TILE][TILEX];

	int Row = by * TILEY + ty;
	int Col = bx * TILEX + tx;
	float Pvalue = 0;

	for (int i = 0; i < n / TILE; ++i) {

		if (TILE <= TILEX){
			if (tx < TILE)
				mad[ty][tx] = ad[Row * n + i * TILE + tx];
		}
		else
			for (int j = 0; j < TILE/TILEX; ++j)
				mad[ty][tx + j*TILEX] = ad[Row * n + ((TILE/TILEX)*i + j) * TILEX + tx];

		if (TILE <= TILEY){
			if (ty < TILE)
				mbd[ty][tx] = bd[(i * TILE + ty) * n + Col];
		}	
		else	
			for (int j = 0; j < TILE / TILEY; ++j) 
				mbd[ty + j * TILEY][tx] = bd[(((TILE / TILEY) * i + j) * TILEY + ty) * n + Col];
				

		__syncthreads();

		for (int k = 0; k < TILE; ++k) {
			Pvalue += mad[ty][k] * mbd[k][tx];
		}
		__syncthreads();
	}
	cd[Row * n + Col] = Pvalue;
}
