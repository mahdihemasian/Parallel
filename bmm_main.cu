#include "hip/hip_runtime.h"
//Do NOT MODIFY THIS FILE

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "gputimer.h"
#include "gpuerrors.h"
#include "bmm.h"

// ===========================> Functions Prototype <===============================
void fill(float* data, int size);
double calc_mse(float* data1, float* data2, int size);
void cpuKernel_yx(const float* const a, const float* const b, float* c, const int m, const int n, const int y, const int x);
void cpuKernel_y(const float* const a, const float* const b, float* c, const int m, const int n, const int y);
void cpuKernel(const float* const a, const float* const b, float* c, const int m, const int n);
void gpuKernel(const float* const a, const float* const b, float* c, const int m, const int n, double* gpu_kernel_time);
// =================================================================================

int main(int argc, char** argv) {

    struct hipDeviceProp_t p;
    hipGetDeviceProperties(&p, 0);
    printf("Device Name: %s\n", p.name);
	
	// get parameter from command line to build Matrix dimension
	// check for 10<=m<=13, because m>=14 do not fit in the memory of our GPU, i.e., 1GB.
	int m = atoi(argv[1]);
    	int n = (1 << m);
	
	// allocate memory in CPU for calculation
	float* a;
	float* b;
	float* c_serial;
	float* c;
	a        = (float*)malloc(n*n * sizeof(float));
	b        = (float*)malloc(n*n * sizeof(float));
	c_serial = (float*)malloc(n*n * sizeof(float));
	c        = (float*)malloc(n*n * sizeof(float));
	
	// fill a, b matrices with random values between -16.0f and 16.0f
	srand(0);
	fill(a, n*n);
	fill(b, n*n);

	// CPU calculations
	if (m<=10) {
		cpuKernel (a, b, c_serial, m, n);
	} else {
		cpuKernel_y (a, b, c_serial, m, n, 0);   // first row
		cpuKernel_y (a, b, c_serial, m, n, n-1); // last row
	}
		
	// GPU calculations
	double gpu_kernel_time = 0.0;
	clock_t t1 = clock(); 
	gpuKernel (a, b, c, m, n, &gpu_kernel_time);
    clock_t t2 = clock(); 
		
	// check correctness of GPU calculations against CPU
	double mse = 0.0;
	if (m<=10) {
		mse += calc_mse( c_serial, c, n*n );
	} else {
		mse += calc_mse( c_serial          , c          , n ); // first row
		mse += calc_mse( c_serial + n*(n-1), c + n*(n-1), n ); // last row
	}

	printf("m=%d n=%d GPU=%g ms GPU-Kernel=%g ms mse=%g\n",
	m, n, (t2-t1)/1000.0, gpu_kernel_time, mse);
		
	// free allocated memory for later use
	free(a);
	free(b);
	free(c_serial);
	free(c);
   
	return 0;
}
//-----------------------------------------------------------------------------
void fill(float* data, int size) {
    for (int i=0; i<size; ++i)
        data[i] = (float) (rand() % 17 - 8);
}

double calc_mse (float* data1, float* data2, int size) {
	double mse = 0.0;
	int i; for (i=0; i<size; i++) {
		double e = data1[i]-data2[i];
		e = e * e;
		mse += e;
	}
	return mse;
}
//-----------------------------------------------------------------------------
void cpuKernel_yx(const float* const a, const float* const b, float* c, const int m, const int n, 
                  const int y, const int x) { // one element: y,x
	mem2d(c,m,y,x)=0.0f;
    for(int k=0; k<n; k++) {
		mem2d(c,m,y,x) += mem2d(a,m,y,k) * mem2d(b,m,k,x);
	}
}
void cpuKernel_y(const float* const a, const float* const b, float* c, const int m, const int n,
                 const int y) { // one row: y
    for(int x=0; x<n; x++) {
		cpuKernel_yx(a,b,c,m,n,y,x);
	}
}
void cpuKernel(const float* const a, const float* const b, float* c, const int m, const int n) { // entire matrix
    for(int y=0; y<n; y++)
    for(int x=0; x<n; x++) {
		cpuKernel_yx(a,b,c,m,n,y,x);
	}
}
//-----------------------------------------------------------------------------
void gpuKernel(const float* const a, const float* const b, float* c, const int m, const int n, double* gpu_kernel_time) {

	float* ad;
	float* bd;
	float* cd;

    HANDLE_ERROR(hipMalloc((void**)&ad, n*n * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&bd, n*n * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&cd, n*n * sizeof(float)));

    HANDLE_ERROR(hipMemcpy(ad, a, n*n * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(bd, b, n*n * sizeof(float), hipMemcpyHostToDevice));

	dim3 dimGrid = getDimGrid(m,n); //modify this function in bmm.cu
	dim3 dimBlock = getDimBlock(m,n); //modify this function in bmm.cu

	GpuTimer timer;
    timer.Start();
	kernelFunc<<< dimGrid,dimBlock >>>(ad, bd, cd, m, n); //modify this function in bmm.cu
	timer.Stop();
	*gpu_kernel_time = timer.Elapsed();
    
	HANDLE_ERROR(hipMemcpy(c, cd, n*n * sizeof(float), hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(ad));
    HANDLE_ERROR(hipFree(bd));
    HANDLE_ERROR(hipFree(cd));
}
