#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "fft.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

//-----------------------------------------------------------------------------

__global__ void sort_rad4(float* input, float* input1, unsigned int M, int k) 
{

    int i = (bz*gridDim.y*gridDim.x + by * gridDim.x + bx) * blockDim.x  + tx;

    unsigned int j = i;
    j = ((j & 0xcccccccc) >> 2) | ((j & 0x33333333) << 2);
    j = ((j & 0xf0f0f0f0) >> 4) | ((j & 0x0f0f0f0f) << 4);
    j = ((j & 0xff00ff00) >> 8) | ((j & 0x00ff00ff) << 8);
    j = (j >> 16) | (j << 16);
    j >>= 32-M;

    input[j]=input1[i + k];
    
}


__global__ void fft_rad2 (float* x_r_d, float* x_i_d ,const unsigned int N, int M){

    int i = (bz*gridDim.y*gridDim.x + by * gridDim.x + bx) * blockDim.x  + tx;

    float angel = -1 * 2 * PI * ((i*(N/(2*M)))-(i/M)*(N/2)) / N;
	float Wr = cos(angel);                   
	float Wi = sin(angel);

	float x1_r, x2_r, x1_i, x2_i;
	
	x1_r = x_r_d[i+(i/M)*M];         
	x2_r = x_r_d[i+(i/M)*M+(M)];
	
	x1_i = x_i_d[i+(i/M)*M];
	x2_i = x_i_d[i+(i/M)*M+(M)];
   
	x_r_d[i+(i/M)*M] = x1_r + Wr * x2_r - Wi * x2_i;
	x_i_d[i+(i/M)*M] = x1_i + Wr * x2_i + Wi * x2_r;
	
	x_r_d[i+(i/M)*M+(M)] = x1_r - Wr * x2_r + Wi * x2_i;
	x_i_d[i+(i/M)*M+(M)] = x1_i - Wr * x2_i - Wi * x2_r;		
		
}


__global__ void fft_rad4 (float* x_r_d, float* x_i_d, const unsigned int N, int M,unsigned int k)
{

    int i = (bz*gridDim.y*gridDim.x + by * gridDim.x + bx) * blockDim.x + tx;

	float x1_r, x2_r, x1_i, x2_i, x4_r, x3_r, x4_i, x3_i;	
	float y2_i, y3_i, y4_i, y2_r, y3_r, y4_r;
	float teta  = -2*PI*(i%M) / (M*4);	
	
	
	x1_r = x_r_d[(i/M)*(4*M)+(i%M) + k];      
	x2_r = x_r_d[(i/M)*(4*M)+(i%M) + M + k];
	x3_r = x_r_d[(i/M)*(4*M)+(i%M) + 2*M + k];
	x4_r = x_r_d[(i/M)*(4*M)+(i%M) + 3*M + k];
	
	x1_i = x_i_d[(i/M)*(4*M)+(i%M) + k];
	x2_i = x_i_d[(i/M)*(4*M)+(i%M) + M + k];
	x3_i = x_i_d[(i/M)*(4*M)+(i%M) + 2*M + k];
	x4_i = x_i_d[(i/M)*(4*M)+(i%M) + 3*M + k];	
	
	float aaa = cos(teta);
	float bbb = sin(teta);

	y2_r = x2_r * aaa - x2_i * bbb;
	y2_i = x2_r * bbb + x2_i * aaa;

    aaa = cos(2*teta);
	bbb = sin(2*teta);

	y3_r = x3_r * aaa - x3_i * bbb;
	y3_i = x3_r * bbb + x3_i * aaa;

    aaa = cos(3*teta);
	bbb = sin(3*teta);

	y4_r = x4_r * aaa - x4_i * bbb;
	y4_i = x4_r * bbb + x4_i * aaa;	
	
	
	x_r_d[(i/M)*(M*4)+(i%M) + k] = x1_r + y2_r + y3_r + y4_r;
	x_i_d[(i/M)*(M*4)+(i%M) + k] = x1_i + y2_i + y3_i + y4_i;
	
	x_r_d[(i/M)*(M*4)+(i%M) + M + k] = x1_r + y2_i - y3_r - y4_i;
	x_i_d[(i/M)*(M*4)+(i%M) + M + k] = x1_i - y2_r - y3_i + y4_r;
	
	x_r_d[(i/M)*(M*4)+(i%M) + 2*M + k] = x1_r - y2_r + y3_r - y4_r;
	x_i_d[(i/M)*(M*4)+(i%M) + 2*M + k] = x1_i - y2_i + y3_i - y4_i;
	
	x_r_d[(i/M)*(M*4)+(i%M) + 3*M + k] = x1_r - y2_i - y3_r + y4_i;
	x_i_d[(i/M)*(M*4)+(i%M) + 3*M + k] = x1_i + y2_r - y3_i - y4_r;
	
}


__global__ void transfer(float* x, float* temp, float* x1, float* temp1) {

	int i = (bz*gridDim.y*gridDim.x + by * gridDim.x + bx) * blockDim.x  + tx;

	x[i] = temp[i];
	x1[i] = temp1[i];

}


__global__ void transfer1(float* x, float* temp, int k) {

	int i = (bz*gridDim.y*gridDim.x + by * gridDim.x + bx) * blockDim.x  + tx;

	x[i + k] = temp[i];
}


__global__ void transpose(float* x, float* tmp, float* x1, float* tmp1, const unsigned int N) 
{
    int i = (bz*gridDim.y*gridDim.x + by * gridDim.x + bx) * blockDim.x  + tx;

    if(i % 2 == 0)
    {
        tmp[i/2] = x[i];
        tmp1[i/2] = x1[i];
    }
    else
    {
        tmp[i/2+N/2] = x[i];
        tmp1[i/2+N/2] = x1[i];
    }

}


void sort_even_number(float* x_r_d, float* x_i_d ,const unsigned int N,const unsigned int M, int k)
{
    float* tmp;

    hipMalloc((void**)&tmp, sizeof(float) * N);

    dim3 dimGrid((N / (512*512)), 32, 32);
	dim3 dimBlock(256, 1, 1);

    sort_rad4 <<< dimGrid, dimBlock >>>(tmp, x_r_d, M, k);
    transfer1 <<< dimGrid, dimBlock >>>(x_r_d,tmp, k);
    sort_rad4 <<< dimGrid, dimBlock >>>(tmp, x_i_d, M, k);
    transfer1 <<< dimGrid, dimBlock >>>(x_i_d,tmp, k);

    hipFree(tmp);

}


void sort_odd_number(float* x_r_d, float* x_i_d ,const unsigned int N,const unsigned int M)
{
    float* tmp_r;
    float* tmp_i;

    hipMalloc((void**)&tmp_r, sizeof(float) * N);
    hipMalloc((void**)&tmp_i, sizeof(float) * N);

    dim3 dimGrid(N/1024, 1, 1);
	dim3 dimBlock(1024, 1, 1);
    transpose<<<dimGrid , dimBlock>>>(x_r_d, tmp_r, x_i_d, tmp_i, N);
    transfer<<<dimGrid , dimBlock>>>(x_r_d, tmp_r, x_i_d, tmp_i);

    hipFree(tmp_r);
    hipFree(tmp_i);

    sort_even_number(x_r_d, x_i_d, N/2, M-1, 0);
    sort_even_number(x_r_d, x_i_d, N/2, M-1, N/2);

}



void gpuKernel(float* x_r_d, float* x_i_d, const unsigned int N, const unsigned int M)
{
 	
	if(M%2 == 0)
    {
        sort_even_number(x_r_d, x_i_d, N, M, 0);

        dim3 dimGrid((N / (16*256)), 8, 1);
	    dim3 dimBlock(128, 1, 1);

	    for (int i=1; i<N; i*=4)  
	    {
	        fft_rad4 <<< dimGrid, dimBlock >>>(x_r_d, x_i_d, N, i, 0);
		}

    }

    else
    {
	    sort_odd_number(x_r_d, x_i_d, N, M);

        dim3 dimGrid((N / (32*256)), 8, 1);
	    dim3 dimBlock(128, 1, 1);

        for (int i=1; i<N/2; i*=4)  
	    {
	        fft_rad4 <<< dimGrid, dimBlock >>>(x_r_d, x_i_d, N, i, 0);
	        fft_rad4 <<< dimGrid, dimBlock >>>(x_r_d, x_i_d, N, i, N/2);
		}

        dim3 dimGrid1((N / (1024*256*2)), 32, 32);
	    dim3 dimBlock1(256, 1, 1);
        fft_rad2 <<< dimGrid1, dimBlock1 >>>(x_r_d, x_i_d, N, N/2);
        
    }

	
}