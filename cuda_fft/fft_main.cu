#include "hip/hip_runtime.h"
//Do NOT MODIFY THIS FILE

#include "fft.h"

// ===========================> Functions Prototype <===============================
void fill(float* data, int size);
double calc_mse(float* data1_r, float* data1_i, float* data2_r, float* data2_i, int size);
void get_inputs(int argc, char *argv[], unsigned int& N, unsigned int& M);
void cpuKernel(float* X_serial_r, float* X_serial_i, int n, float* tmp_r, float* tmp_i);
void gpuKernels(float* x_r, float* x_i, float* X_r, float* X_i, unsigned int N, unsigned int M, double* gpu_kernel_time);
// =================================================================================

int main(int argc, char *argv[]) {

    struct hipDeviceProp_t p;
    hipGetDeviceProperties(&p, 0);
    printf("Device Name: %s\n", p.name);

    // get parameters from command line
    unsigned int N, M;
    get_inputs(argc, argv, N, M);

    // allocate memory in CPU for calculation
    float* x_r; // real part
    float* x_i; // imaginary part
    float* X_serial_r;
    float* X_serial_i;
    float* X_r;
    float* X_i;
    x_r = (float*) malloc(N * sizeof(float));
    x_i = (float*) malloc(N * sizeof(float));
    X_serial_r = (float*) malloc(N * sizeof(float));
    X_serial_i = (float*) malloc(N * sizeof(float));
    X_r = (float*) malloc(N * sizeof(float));
    X_i = (float*) malloc(N * sizeof(float));

    // fill x_r and x_i arrays with random values between -8.0f and 8.0f
    srand(0);
    fill(x_r, N);
    fill(x_i, N);
	int i; for (i = 0; i < N; i++) {
		X_serial_r[i] = x_r[i];
		X_serial_i[i] = x_i[i];
	}

    // time measurement for CPU calculation
	float *tmp_r, *tmp_i;
	tmp_r = (float*) malloc(N * sizeof(float));
    tmp_i = (float*) malloc(N * sizeof(float));
    clock_t t0 = clock();
    cpuKernel(X_serial_r, X_serial_i, N, tmp_r, tmp_i);
    clock_t t1 = clock();
	free(tmp_r); free(tmp_i);

    // time measurement for GPU calculation
	double gpu_kernel_time = 0.0;
    clock_t t2 = clock();
	gpuKernels(x_r, x_i, X_r, X_i, N, M, &gpu_kernel_time);
    clock_t t3 = clock();

    // check correctness of calculation
    double mse = calc_mse(X_serial_r, X_serial_i, X_r, X_i, N);
	printf("m=%d n=%d CPU=%g ms GPU=%g ms GPU-Kernels=%g ms mse=%g\n",
	M, N, (t1-t0)/1000.0, (t3-t2)/1000.0, gpu_kernel_time, mse);
	
	/*
	for (i = 0; i<N; i++) {
		printf("%f\t%f\n", x_r[i], x_i[i]);
	}
	printf("\n");
	for (i = 0; i<N; i++) {
		printf("%f\t%f\n", X_serial_r[i], X_serial_i[i]);
	}
	*/
	
    // free allocated memory for later use
    free(x_r);
    free(x_i);
    free(X_serial_r);
    free(X_serial_i);
    free(X_r);
    free(X_i);

    return 0;
}

//-----------------------------------------------------------------------------
void gpuKernels(float* x_r, float* x_i, float* X_r, float* X_i, unsigned int N, unsigned int M, double* gpu_kernel_time) {
    float* x_r_d;
    float* x_i_d;
    //float* X_r_d;
    //float* X_i_d;

    HANDLE_ERROR(hipMalloc((void**)&x_r_d, N * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&x_i_d, N * sizeof(float)));
    //HANDLE_ERROR(hipMalloc((void**)&X_r_d, N * sizeof(float)));
    //HANDLE_ERROR(hipMalloc((void**)&X_i_d, N * sizeof(float)));

    HANDLE_ERROR(hipMemcpy(x_r_d, x_r, N * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(x_i_d, x_i, N * sizeof(float), hipMemcpyHostToDevice));

	GpuTimer timer;
    timer.Start();
	gpuKernel(x_r_d, x_i_d, N, M);/*<<<dim3(32,1,1),dim3(32,1,1)>>>(x_r_d, x_i_d, N, M);*/
	timer.Stop();
	*gpu_kernel_time = timer.Elapsed();
	
    //HANDLE_ERROR(hipMemcpy(X_r, X_r_d, N * sizeof(float), hipMemcpyDeviceToHost));
    //HANDLE_ERROR(hipMemcpy(X_i, X_i_d, N * sizeof(float), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(X_r, x_r_d, N * sizeof(float), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(X_i, x_i_d, N * sizeof(float), hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(x_r_d));
    HANDLE_ERROR(hipFree(x_i_d));
    //HANDLE_ERROR(hipFree(X_r_d));
    //HANDLE_ERROR(hipFree(X_i_d));
}
//-----------------------------------------------------------------------------
void cpuKernel(float* X_serial_r, float* X_serial_i, int n, float* tmp_r, float* tmp_i) {
	if(n > 1) {	// otherwise, do nothing and return
		int k, m;
		float z_r, z_i, w_r, w_i;
		float *vo_r, *vo_i, *ve_r, *ve_i;
		ve_r = tmp_r; ve_i = tmp_i;
		vo_r = tmp_r + n/2; vo_i = tmp_i + n/2;
		
		for(k=0; k<n/2; k++) {
			ve_r[k] = X_serial_r[2*k]; ve_i[k] = X_serial_i[2*k];
			vo_r[k] = X_serial_r[2*k+1]; vo_i[k] = X_serial_i[2*k+1];
		}
		cpuKernel(ve_r, ve_i, n/2, X_serial_r, X_serial_i);	// FFT on even-indexed elements of v[]
		cpuKernel(vo_r, vo_i, n/2, X_serial_r, X_serial_i);	// FFT on odd-indexed elements of v[]
		
		for(m=0; m<n/2; m++) {
			w_r =  cos((2*PI*m)/n);
			w_i = -sin((2*PI*m)/n);
			z_r = w_r*vo_r[m] - w_i*vo_i[m];	// Re(w*vo[m])
			z_i = w_r*vo_i[m] + w_i*vo_r[m];	// Im(w*vo[m])
			X_serial_r[  m  ] = ve_r[m] + z_r;
			X_serial_i[  m  ] = ve_i[m] + z_i;
			X_serial_r[m+n/2] = ve_r[m] - z_r;
			X_serial_i[m+n/2] = ve_i[m] - z_i;
		}
	}
	return;
}
//-----------------------------------------------------------------------------
void get_inputs(int argc, char *argv[], unsigned int& N, unsigned int& M)
{
    if (
	argc != 2 || 
	atoi(argv[1]) < 0 || atoi(argv[1]) > 26 
	) {
        printf("<< Error >>\n");
        printf("Enter the following command:\n");
        printf("\t./a.out  M\n");
        printf("\t\tM must be between 0 and 26\n");
		exit(-1);
    }
	M = atoi(argv[1]);
    N = (1 << M);
}
//-----------------------------------------------------------------------------
void fill(float* data, int size) {
    for (int i = 0; i < size; i++)
        data[i] = (float)(rand() % 17 - 8);
}
double calc_mse(float* data1_r, float* data1_i, float* data2_r, float* data2_i, int size) {
    double mse = 0.0;
    int i;
    for (i = 0; i < size; i++) {
        double e_r = data1_r[i] - data2_r[i];
        double e_i = data1_i[i] - data2_i[i];
        double e = e_r * e_r + e_i * e_i;
        mse += e;
    }
    return mse/size;
}
